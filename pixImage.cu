#include "hip/hip_runtime.h"
/**
 * @file pixImage.cu
 * @author Kevin Xie (kevinx) 
 *         Anthony Meza (abmeza)
 * @brief Implementation of PixImage. Also a replica off pixImage.cpp, but 
 *        created in order to implement cuda. Some of the structure was 
 *        inspired by assignment 2 in 15418 which also used cuda to render
 *        circles on a grid.
 *
 * @note  On GHC machine run the following to get cuda working:
 *        export PATH=/usr/local/depot/cuda/bin:${PATH}
 *        export LD_LIBRARY_PATH=/usr/local/depot/cuda/lib64/:${LD_LIBRARY_PATH}
 * 
 * @version 0.1
 * @date 2022-04-22
 * 
 * @copyright Copyright (c) 2022
 * 
 */

// Constants to regulate what we print
//#define DEBUG     // misc. debug statements
#define RUN_DEBUG // debug statements that check running progress
#define TIMING // Calculate and print timing information


// Import util libraries
#include "CycleTimer.h"
#include "pixImage.h"


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <algorithm> 
#include <stack>
#include <cmath>
using namespace std;


//********************************************************//
//*******************  GLOBAL VARIABLES ******************//
//********************************************************//

// storage for global constants that usble by GPU
struct GlobalConstants {
    // Input Image Content 
    int in_width, in_height;  //<- width and height of input_img, gives pixel dimensions
    int M_pix;                //<- # of pixels from the input image (M from paper)
    unsigned char *input_img; //<- input image loaded, uses rgb values for pixels (0-255)
    LabColor *input_img_lab;  //<- input image, using cielab values 

    // Output Image 
    int out_width, out_height; //<- output version of width, height
    int N_pix;                 //<- # of pixels in the output image (N from paper)
    unsigned char *output_img; //<- output version of input_img
    unsigned char *spoutput_img; //<- debug output for superpixels
    LabColor *buf_lab;         //<- buffer for smoothing and palette refinement

    // Superpixel calculation 
    FloatVec *superPixel_pos; //<- Super pixel coordinate positions "on input image"
    LabColor *sp_mean_lab;  //<- superpixel mean color value
    int *region_map;      //<- array with values for pixels associated with a specific superpixel 

    // Palette 
    int K_colors;          //<- number of colors we aim to use in the pallette
    int * palette_size;      //<- POINTER SO WE CAN MODIFY Current # of colors stored in palette_lab
    PalettePair *palette_pairs;
    int *palette_assign; //<- palette assignment for each superpixel
    LabColor *palette_lab; //<- palette array with color values in palette
    LabColor *average_palette; //<- average palette array with average color values
   
    bool *palette_complete; //<- POINTER SO WE CAN MODIFY

    float *prob_c;         //<- array of probabiities that a color in the palette is set to ANY super pixel
    float prob_sp;         //<- array of probabiities of each super pixel TODO:EDIT
    float *prob_c_if_sp;   //<- List of P(c_k|p_s) values for all superpixels
    
    // Temperature
    float *T;   //<- POINTER SO WE CAN MODIFY Current temperature

    //extra
    bool *converged;
    float S;

};

// Constant for GPU
__constant__ GlobalConstants cuGlobalConsts;


//*********************************************************************//
//******************* INLINE KERNEL HELPER FUNCTIONS ******************//
//*********************************************************************//

/**
 * @brief max function
 * 
 * @param item1 
 * @param item2 
 * @return int 
 */
__device__ __inline__ int cuDevMax(int item1, int item2) {
    return (item1 >= item2) ? item1 : item2;
}

/**
 * @brief min function
 * 
 * @param item1 
 * @param item2 
 * @return int
 */
__device__ __inline__ int cuDevMin(int item1, int item2) {
    return (item1 >= item2) ? item2 : item2;
}



__device__ __inline__ int cuDevMaxEigen3(float *matrix, float *value, LabColor *vector) {
    
    // extract unique values from top triangle
    float a = matrix[0];
    float b = matrix[4];
    float c = matrix[8];
    float d = matrix[1];
    float e = matrix[5];
    float f = matrix[3];

    float x_1 = a*a + b*b + c*c - a*b - a*c - b*c + 3*(d*d + f*f + e*e);
    float x_2 = -(2*a - b - c) * (2*b - a - c) * (2*c - a - b) + 
                9*((2*c - a - b)*d*d + (2*b - a - c)*f*f + (2*a - b - c)*e*e) - 54*d*e*f;
    
    float phi;
    if (x_2 > 0) {
        phi = atan(sqrt(4*(x_1*x_1*x_1) - x_2*x_2)/x_2);
    } else if (x_2 == 0) {
        phi = M_PI/2;
    } else {
        phi = atan(sqrt(4*(x_1*x_1*x_1) - x_2*x_2)/x_2) + M_PI;
    }

    float lam_1 = (a + b + c - 2*sqrt(x_1)*cos(phi/3))/3;
    float lam_2 = (a + b + c + 2*sqrt(x_1)*cos((phi - M_PI)/3))/3;
    float lam_3 = (a + b + c + 2*sqrt(x_1)*cos((phi + M_PI)/3))/3;

    float lam = (lam_2 > lam_1) ? lam_2 : ((lam_3 > lam_1) ? lam_3 : lam_1);

    // unlikely special case causes divide by zero
    if (f == 0 || f*(b-lam)-d*e == 0) {
        printf("(f: %f, b: %f, lam: %f, d: %f, e: %f)\n", f, b, lam, d, e);
        return -1;
    }

    // store highest eigenvalue
    * value = lam;

    float m = (d*(c-lam)-e*f)/(f*(b-lam)-d*e);

    float v_0 = (lam - c - e*m)/f;

    // store associated eigenvector
    vector->L = v_0;
    vector->a = m;
    vector->b = 1.f;

    return 0;
}


__device__ __inline__ float cuDevDist_k(int m, float S, float l_k, float a_k, float b_k, int x_k, int y_k,
           float l_i, float a_i, float b_i, int x_i, int y_i) {
    float d_lab = sqrt(pow(l_k - l_i, 2.f) + pow(a_k - a_i, 2.f) + pow(b_k - b_i, 2.f));
    float d_xy = sqrt(pow((float) x_k - x_i, 2.f) + pow((float) y_k - y_i, 2.f));
    float k = ((float) m) / S;
    return d_lab + k * d_xy;
}


__device__ __inline__ float cuDevGaussian(float x, float sigma, float mean) {
    return exp((x-mean)*(x-mean)/(-2.0f*sigma*sigma))/sqrt(6.28319*sigma*sigma);
}


/** https://www.image-engineering.de/library/technotes/958-how-to-convert-between-srgb-and-ciexyz **/
__device__ __inline__  float cuDevLinearize(int V) {
    float Vf = ((float) V) / 255.0f;
    if (Vf > rgbT_32f) {
        Vf = (Vf + rgbLShift_32f)*rgbLScale_32f;
        return pow(Vf, rgbLPow_32f) * 100;
    } else {
        return (Vf * rgbScale_32f) * 100;
    }
}

__device__ __inline__ unsigned char cuDevDelin(float V) {
    V = V / 100.f;
    float Vf;
    if (V > rgbTinv_32f) {
        Vf = rgbInvScale_32f * pow(V, rgbLPowInv_32f) - rgbLShift_32f;
    } else {
        Vf = rgbSmallScale_32f * V; 
    }
    //Vf = (Vf < 0) ? 0 : Vf;
    return (unsigned char) roundf(Vf * 255.f);
}

__device__ __inline__ float cuDevRGBtoX(float R, float G, float B) {
    return labXr_32f * R + labXg_32f * G + labXb_32f * B;
}

__device__ __inline__ float cuDevRGBtoY(float R, float G, float B) {
    return labYr_32f * R + labYg_32f * G + labYb_32f * B;
}

__device__ __inline__ float cuDevRGBtoZ(float R, float G, float B) {
    return labZr_32f * R + labZg_32f * G + labZb_32f * B;
}

/**
 * @brief Peicewise function to help calculate conversion XYZ to Lab
 *   
 * @param t 
 * @return float output of function
 */
__device__ __inline__ float cuDevF_func(float t){
    if (t > lab_delta3_32f) {
        return cbrt(t);
    } else {
        return t*labSmallScale_32f + labSmallShift_32f;
    }
}

__device__ __inline__ float cuDevXYZtoL(float X, float Y, float Z) {
    Y = cuDevF_func(Y);

    return Y*labLScale_32f - labLShift_32f;

}

__device__ __inline__ float cuDevXYZtoA(float X, float Y, float Z) {
    X = cuDevF_func(X);
    Y = cuDevF_func(Y);
    
    return 500.f*(X - Y);
}

__device__ __inline__ float cuDevXYZtoB(float X, float Y, float Z) {
    Y = cuDevF_func(Y);
    Z = cuDevF_func(Z);

    return 200.f*(Y - Z);
}

__device__ __inline__ float cuDevXYZtoR(float X, float Y, float Z) {
    return labRx_32f * X + labRy_32f * Y + labRz_32f * Z;
}

__device__ __inline__ float cuDevXYZtoG(float X, float Y, float Z) {
    return labGx_32f * X + labGy_32f * Y + labGz_32f * Z;

}

__device__ __inline__ float cuDevXYZtorgB(float X, float Y, float Z) {
    return labBx_32f * X + labBy_32f * Y + labBz_32f * Z;
}

/* http://www.brucelindbloom.com/index.html?Eqn_Lab_to_XYZ.html */
/**
 * @brief Peicewise function to help calculate conversion Lab to XYZ
 *   
 * @param t 
 * @return float output of function
 */
__device__ __inline__ float cuDevFInv_func(float t){
    if ( t > lab_delta_32f) {
        return pow(t, labPow_32f);
    } else {
        return labSmallScaleInv_32f*(t - labSmallShift_32f);
    }
}

__device__ __inline__ float cuDevLABtoX(float L, float a, float b) {
    float fy = (L + labLShift_32f)/labLScale_32f;
    float fx = fy + a*labAScale_32f;
    
    return cuDevFInv_func(fx);
}

__device__ __inline__ float cuDevLABtoY(float L, float a, float b) {
    float fy = (L + labLShift_32f)/labLScale_32f;

    return cuDevFInv_func(fy);
}

__device__ __inline__ float cuDevLABtoZ(float L, float a, float b) {
    float fy = (L + labLShift_32f)/labLScale_32f;
    float fz = fy - b*labBScale_32f;

    return cuDevFInv_func(fz);
}

__device__ __inline__ void cuDevrgb2lab(int R, int G, int B, float *L, float *a, float *b) {
    // printf("_RGB2LAB_\n");
    // printf("(R:%d,G:%d,B:%d)->",R,G,B);
    float Rf = cuDevLinearize(R);
    float Gf = cuDevLinearize(G);
    float Bf = cuDevLinearize(B);
    
    // printf("(Rf:%f,Bf:%f,Gf:%f)->",Rf,Gf,Bf);

    float X = cuDevRGBtoX(Rf, Gf, Bf);
    float Y = cuDevRGBtoY(Rf, Gf, Bf);
    float Z = cuDevRGBtoZ(Rf, Gf, Bf);
    
    // printf("(X:%f,Y:%f,Z:%f)->",X,Y,Z);

    X *= labXScale_32f;
    Y *= labYScale_32f;
    Z *= labZScale_32f;
    
    // printf("(XS:%f,YS:%f,ZS:%f)->",X,Y,Z);

    *L = cuDevXYZtoL(X, Y, Z);
    *a = cuDevXYZtoA(X, Y, Z);
    *b = cuDevXYZtoB(X, Y, Z);
    
    // printf("(L:%f,a:%f,b:%f)\n",*L,*a,*b);
}

__device__ __inline__ void cuDevlab2rgb(float L, float a, float b, unsigned char *R, unsigned char *G, unsigned char *B) {
    // printf("_LAB2RGB_\n");
    // printf("(L:%f,a:%f,b:%f)->",L,a,b);
    float X = cuDevLABtoX(L, a, b);
    float Y = cuDevLABtoY(L, a, b);
    float Z = cuDevLABtoZ(L, a, b);
    
    // printf("(XS:%f,YS:%f,ZS:%f)->",X,Y,Z);
    X *= labXScaleInv_32f;
    Y *= labYScaleInv_32f;
    Z *= labZScaleInv_32f;

    // printf("(X:%f,Y:%f,Z:%f)->",X,Y,Z);

    float Rf = cuDevXYZtoR(X, Y, Z);
    float Gf = cuDevXYZtoG(X, Y, Z);
    float Bf = cuDevXYZtorgB(X, Y, Z);

    // printf("(Rf:%f,Bf:%f,Gf:%f)->",Rf,Gf,Bf);

    *R = cuDevDelin(Rf);
    *G = cuDevDelin(Gf);
    *B = cuDevDelin(Bf);
    // printf("(R:%d,G:%d,B:%d)",*R,*G,*B);
}


//***************************************************************//
//*******************  INLINE KERNEL FUNCTIONS ******************//
//***************************************************************//

__device__ __inline__ void getMajorAxis(int palette_index, float *value, LabColor *vector) {
    // *** TODO TRANSFER OVER CONSTANTS ***//
    int N_pix = cuGlobalConsts.N_pix;
    int out_height = cuGlobalConsts.out_height;
    int out_width = cuGlobalConsts.out_width;
    float prob_sp = cuGlobalConsts.prob_sp;

    float *prob_c_if_sp = cuGlobalConsts.prob_c_if_sp;
    float *prob_c = cuGlobalConsts.prob_c;
    LabColor *palette_lab = cuGlobalConsts.palette_lab;
    LabColor *sp_mean_lab = cuGlobalConsts.sp_mean_lab;
    
    float covariance[9];
    memset(covariance, 0, 9*sizeof(float));
    float sum = 0;

    // compute covariance matrix
    for (int j = 0; j < out_height; j++) {
        for (int i = 0; i < out_height; i++) {
            int idx = j*out_width + i;

            // probability of superpixel given palette color
            float prob_oc = prob_c_if_sp[palette_index * N_pix + idx] 
                            * prob_sp / prob_c[palette_index];
            sum += prob_oc;

            // find color error with current superpixel
            LabColor pl_color = palette_lab[palette_index];    
            LabColor sp_color = sp_mean_lab[idx];
            float L_error = fabs(pl_color.L - sp_color.L);
            float a_error = fabs(pl_color.a - sp_color.a);
            float b_error = fabs(pl_color.b - sp_color.b);

            // update covariance
            covariance[0] += prob_oc*L_error*L_error;
            covariance[1] += prob_oc*a_error*L_error;
            covariance[2] += prob_oc*b_error*L_error;
            covariance[3] += prob_oc*L_error*a_error;
            covariance[4] += prob_oc*a_error*a_error;
            covariance[5] += prob_oc*b_error*a_error;
            covariance[6] += prob_oc*L_error*b_error;
            covariance[7] += prob_oc*a_error*b_error;
            covariance[8] += prob_oc*b_error*b_error;
        }
    }

    LabColor eVec;
    float eVal;

    int error = cuDevMaxEigen3(covariance, &eVal, &eVec);
    if (error < 0) {
        printf("maxEigen3 special case\n");
    }

    float len = sqrt(eVec.L*eVec.L + eVec.a*eVec.a + eVec.b*eVec.b);
    if (len > 0) {
        eVec.L *= (1.0f/len);
        eVec.a *= (1.0f/len);
        eVec.b *= (1.0f/len);
    }

    *value = eVal;
    *vector = eVec;
}


__device__ __inline__ void pushPaletteColor(LabColor color, float prob) {
    // *** TODO TRANSFER OVER CONSTANTS ***//
    LabColor *palette_lab = cuGlobalConsts.palette_lab;
    float *prob_c = cuGlobalConsts.prob_c;
    int *palette_size = cuGlobalConsts.palette_size;

    palette_lab[*palette_size] = color;
    prob_c[*palette_size] = prob;
    (*palette_size) = (*palette_size) + 1; 
}

__device__ __inline__ void pushPalettePair(int a, int b) {
    // *** TODO TRANSFER OVER CONSTANTS ***//
    int K_colors = cuGlobalConsts.K_colors;

    PalettePair *palette_pairs = cuGlobalConsts.palette_pairs;
    int *palette_size = cuGlobalConsts.palette_size;

    PalettePair newPair = {a, b};
    int idx = ((*palette_size) >> 1) - 1;
    if (idx < 0 || idx > K_colors) {
        return;
    }
    palette_pairs[idx] = newPair;
}


/**
 * @brief Get the average palette
 */
__device__ __inline__ void inlineGetAveragedPalette() {
    // *** TODO TRANSFER OVER CONSTANTS ***//
    LabColor *average_palette = cuGlobalConsts.average_palette;
    LabColor *palette_lab = cuGlobalConsts.palette_lab;
    int *palette_size = cuGlobalConsts.palette_size;
    PalettePair *palette_pairs = cuGlobalConsts.palette_pairs;
    float *prob_c = cuGlobalConsts.prob_c;
    bool *palette_complete = cuGlobalConsts.palette_complete;


    if (*palette_complete) {
        memcpy(average_palette, palette_lab, (*palette_size) *sizeof(LabColor));
        return;
    }
    for (int i = 0; i < (*palette_size) >> 1; i++) {
        PalettePair pair = palette_pairs[i];
        float weight_a = prob_c[pair.a];
        float weight_b = prob_c[pair.b];
        float total_weight = weight_a + weight_b;
        weight_a /= total_weight;
        weight_b /= total_weight;

        LabColor ca = palette_lab[pair.a];
        LabColor cb = palette_lab[pair.b];

        LabColor avg = {ca.L*weight_a + cb.L*weight_b,
                        ca.a*weight_a + cb.a*weight_b,
                        ca.b*weight_a + cb.b*weight_b};
        
        average_palette[pair.a] = avg;
        average_palette[pair.b] = avg;
    }
}

__device__ __inline__ void inlineCondensePalette() {

    // *** TODO TRANSFER OVER CONSTANTS ***//
    const int N_pix = cuGlobalConsts.N_pix;
    const int K_colors = cuGlobalConsts.K_colors;
    
    LabColor *average_palette = cuGlobalConsts.average_palette;
    int *palette_size = cuGlobalConsts.palette_size;
    bool *palette_complete = cuGlobalConsts.palette_complete;
    PalettePair *palette_pairs = cuGlobalConsts.palette_pairs;
    float *prob_c = cuGlobalConsts.prob_c;
    float *prob_c_if_sp = cuGlobalConsts.prob_c_if_sp;
    int *palette_assign = cuGlobalConsts.palette_assign;
    LabColor *palette_lab = cuGlobalConsts.palette_lab;


    // Allocate local memory
    LabColor *new_palette = new LabColor[K_colors * 2];
    float *new_prob_c = new float[K_colors * 2];
    float *new_prob_c_if_sp = new float[K_colors * 2 * N_pix];
    int *new_palette_assign = new int[N_pix];
    
  

    // for each pair, condense to average
    for(int j = 0; j < (*palette_size) >> 1; ++j) {
        int index_a = palette_pairs[j].a;
        int index_b = palette_pairs[j].b;
        new_palette[j] = average_palette[index_a]; //TODO: SHOULD BE IN KERNALS
        //update the probability of the single superpixel
        new_prob_c[j] = prob_c[index_a] + 
                        prob_c[index_b];

        // reassign superpixels
        for(int i = 0; i < N_pix; i++) {
            new_prob_c_if_sp[j*N_pix + i] = prob_c_if_sp[index_a*N_pix + i];
            if (palette_assign[i] == index_a || palette_assign[i] == index_b) {
                new_palette_assign[i] = j;
            }
        }
    }

    // copy new values
    memcpy(palette_lab, new_palette, K_colors * 2 * sizeof(LabColor));
    memcpy(palette_assign, new_palette_assign, N_pix * sizeof(int));
    
    // TODO: could be wrong?? wtf is prob_oc_
    memcpy(prob_c_if_sp, new_prob_c_if_sp, K_colors * 2 * N_pix * sizeof(float));

    *palette_size = K_colors;
    *palette_complete = true;

    // Delete the stuff
    delete[] new_palette;
    delete[] new_prob_c;
    delete[] new_prob_c_if_sp ;
    delete[] new_palette_assign;
    
    #ifdef RUN_DEBUG
    printf("DONE \n");
    #endif
}

 __device__ __inline__ void splitColor(int pair_index) {
     
    // *** TODO TRANSFER OVER CONSTANTS ***//

    int N_pix = cuGlobalConsts.N_pix;

    int *palette_size = cuGlobalConsts.palette_size;
    PalettePair *palette_pairs = cuGlobalConsts.palette_pairs;
    LabColor *palette_lab = cuGlobalConsts.palette_lab;
    float *prob_c = cuGlobalConsts.prob_c;
    float *prob_c_if_sp = cuGlobalConsts.prob_c_if_sp;


    int index_a = palette_pairs[pair_index].a;
    int index_b = palette_pairs[pair_index].b;

    // perturb a
    LabColor color_a = palette_lab[index_a];
    LabColor color_a_b = color_a;
    LabColor majorAxis;
    float variance;
    getMajorAxis(index_a, &variance, &majorAxis);
    color_a_b.L += majorAxis.L * kSubclusterPertubation;
    color_a_b.a += majorAxis.a * kSubclusterPertubation;
    color_a_b.b += majorAxis.b * kSubclusterPertubation;
    
    // reconstruct pair a and copy probabilities
    prob_c[index_a] *= 0.5f;
    pushPaletteColor(color_a_b, prob_c[index_a]);
    int index_a_b = *palette_size - 1;
    memcpy(&prob_c_if_sp[index_a_b*N_pix], &prob_c_if_sp[index_a*N_pix], N_pix*sizeof(float));
    palette_pairs[pair_index].b = index_a_b;

    // perturb b
    LabColor color_b = palette_lab[index_b];
    LabColor color_b_b = color_b;
    getMajorAxis(index_b, &variance, &majorAxis);
    color_b_b.L += majorAxis.L * kSubclusterPertubation;
    color_b_b.a += majorAxis.a * kSubclusterPertubation;
    color_b_b.b += majorAxis.b * kSubclusterPertubation;

    // reconstruct pair b and copy probabilities
    prob_c[index_b] *= 0.5f;
    pushPaletteColor(color_b_b, prob_c[index_b]);
    int index_b_b = (*palette_size) - 1;
    memcpy(&prob_c_if_sp[index_b_b*N_pix], &prob_c_if_sp[index_b*N_pix], N_pix*sizeof(float));
    pushPalettePair(index_b, index_b_b);
}

//********************************************************//
//*******************  KERNEL FUNCTIONS ******************//
//********************************************************//

/**
 * @brief creates lab version of input_img
 */
__global__ void kernelCreateInputLAB() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //TODO: RUN ON ONE KERNAL FOR NOW
    if (index == 0){
    
    // *** TODO TRANSFER OVER CONSTANTS ***//
    int M_pix = cuGlobalConsts.M_pix;
    unsigned char *input_img = cuGlobalConsts.input_img;
    LabColor *input_img_lab = cuGlobalConsts.input_img_lab;


    unsigned char *p; 
    LabColor *pl;
    for(p = input_img, pl = input_img_lab; p != input_img + (M_pix*3); p += 3, pl ++) 
        cuDevrgb2lab(*p, *(p+1), *(p+2), &(pl->L), &(pl->a), &(pl->b));

    
    }
}
/**
 * @brief kernal that runs initSuperPixels on Device
 */
__global__ void kernelInitSuperPixels() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //TODO: RUN ON ONE KERNAL FOR NOW
    if (index == 0){
    
    // *** TODO TRANSFER OVER CONSTANTS ***//
    int in_width = cuGlobalConsts.in_width;
    int in_height = cuGlobalConsts.in_height;
    int out_width = cuGlobalConsts.out_width;
    int out_height = cuGlobalConsts.out_height;
    FloatVec *superPixel_pos = cuGlobalConsts.superPixel_pos;
    int *region_map = cuGlobalConsts.region_map;


    // Get change in length of values
    float dx = (float) in_width/(float) out_width;
    float dy = (float) in_height/(float) out_height;

    // initialize superpixel positions (centers)
    for (int j = 0; j < out_height; ++j) {
        for (int i = 0; i < out_width; ++i) {

            // Calculate midpoint value
            float x = ((float) i + 0.5f) * dx;
            float y = ((float) j + 0.5f) * dy;
            FloatVec pos =  (FloatVec) {x,y};
            // Set value
            superPixel_pos[out_width * j + i] = pos;


        }
    }

    // Initial assignment of pixels to a specific superpxel  
    for (int j = 0; j < in_height; ++j) {
        for (int i = 0; i < in_width; ++i) {
            // Calculate which superpixel to set
            int x = (int) ((float) i / dx);
            int y = (int) ((float) j / dy);

            // Set Value
            region_map[in_width * j + i] = out_width * y + x;
        }
    }

    }
}
/**
 * @brief kernal that runs updateSuperPixelMeans on Device
 */
__global__ void kernelUpdateSuperPixelMeans() {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //TODO: RUN ON ONE KERNAL FOR NOW
    if (index == 0){
    
    // *** TODO TRANSFER OVER CONSTANTS ***//
    const int N_pix = cuGlobalConsts.N_pix;
    const int in_width = cuGlobalConsts.in_width;
    const int in_height = cuGlobalConsts.in_height;
    const int out_width = cuGlobalConsts.out_width;
    const int out_height = cuGlobalConsts.out_height;

    int *region_map = cuGlobalConsts.region_map;
    LabColor *input_img_lab = cuGlobalConsts.input_img_lab;
    FloatVec *superPixel_pos = cuGlobalConsts.superPixel_pos;
    LabColor *sp_mean_lab = cuGlobalConsts.sp_mean_lab;


    FloatVec *sp_sums = new FloatVec[N_pix];
    LabColor *color_sums = new LabColor[N_pix];
    int *sp_count = new int[N_pix];

    memset(sp_sums, 0, N_pix*sizeof(FloatVec));
    memset(color_sums, 0, N_pix*sizeof(LabColor));
    memset(sp_count, 0, N_pix*sizeof(int));
    // Find the mean colors (from input image) for each superpixel
    for (int j = 0; j < in_height; j++) {
        for (int i = 0; i < in_width; i++) {
            int idx = j*in_width + i;
            int spidx = region_map[idx];
            sp_count[spidx] ++;
            sp_sums[spidx].x += i;
            sp_sums[spidx].y += j;

            color_sums[spidx].L += input_img_lab[idx].L;
            color_sums[spidx].a += input_img_lab[idx].a;
            color_sums[spidx].b += input_img_lab[idx].b;
        }
    }
    
    // Repostion superpixels and update the output color pallete
    for (int j = 0; j < out_height; j++) {
        for (int i = 0; i < out_width; i++) {
            // Index of superpixel
            int spidx = j*out_width + i;

            if (sp_count[spidx] == 0) {
                float dx = (float) in_width/(float) out_width;
                float dy = (float) in_height/(float) out_height;
                float x = ((float) i + 0.5f) * dx;
                float y = ((float) j + 0.5f) * dy;
                sp_mean_lab[spidx] = input_img_lab[((int) round(y))*in_width + ((int) round(x))];
                continue;
            }

            // Calculate new position for super pixel
            float x = sp_sums[spidx].x / sp_count[spidx];
            float y = sp_sums[spidx].y / sp_count[spidx];
            FloatVec newpos = {x, y};
            superPixel_pos[spidx] = newpos;

            // Set output_img_lab to new mean value
            sp_mean_lab[spidx].L = color_sums[spidx].L/sp_count[spidx];
            sp_mean_lab[spidx].a = color_sums[spidx].a/sp_count[spidx];
            sp_mean_lab[spidx].b = color_sums[spidx].b/sp_count[spidx];
        }
    }
    
    delete[] sp_sums; 
    delete[] color_sums;
    delete[] sp_count;

    }
}
/**
 * @brief kernal that runs initializes palette values on Device
 */
__global__ void kernelInitPaletteValues() {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //TODO: RUN ON ONE KERNAL FOR NOW
    if (index == 0){
    

    // *** TODO TRANSFER OVER CONSTANTS ***//
    int N_pix = cuGlobalConsts.N_pix;
    float prob_sp = cuGlobalConsts.prob_sp;

    float *T = cuGlobalConsts.T;
    LabColor *sp_mean_lab = cuGlobalConsts.sp_mean_lab;
    float *prob_c_if_sp = cuGlobalConsts.prob_c_if_sp;


    // Find mean of all M_pix color inputs
    LabColor color_sum = {0.f, 0.f, 0.f};

    //add all colors
    for (int p = 0; p < N_pix; p++) {
        color_sum.L += sp_mean_lab[p].L;
        color_sum.a += sp_mean_lab[p].a;
        color_sum.b += sp_mean_lab[p].b;
    }
    // divide all by M_pix
    color_sum.L = color_sum.L * prob_sp;
    color_sum.a = color_sum.a * prob_sp;
    color_sum.b = color_sum.b * prob_sp;

    #ifdef DEBUG
    printf("color_init: (%f, %f, %f)\n", color_sum.L, color_sum.a, color_sum.b);
    #endif

    // Store color and update prob to any
    pushPaletteColor(color_sum, 0.5f);
    for (int idx = 0; idx < N_pix; idx ++) {
        prob_c_if_sp[idx] = 0.5f;
    }
    LabColor majorAxis;
    float variance;
    getMajorAxis(0, &variance, &majorAxis);
    
    color_sum.L += majorAxis.L * kSubclusterPertubation;
    color_sum.a += majorAxis.a * kSubclusterPertubation;
    color_sum.b += majorAxis.b * kSubclusterPertubation;

    pushPaletteColor(color_sum, 0.5f);
    for (int idx = 0; idx < N_pix; idx ++) {
        prob_c_if_sp[N_pix + idx] = 0.5f;
    }

    pushPalettePair(0, 1);

    *T = sqrt(2*variance) * kT0SafetyFactor;

    }
}

/**
 * @brief Update the average palette
 */
__global__ void kernelGetAveragedPalette() {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //TODO: RUN ON ONE KERNAL FOR NOW
    if (index == 0){
    
    // *** TODO TRANSFER OVER CONSTANTS ***/
    LabColor *average_palette = cuGlobalConsts.average_palette;
    LabColor *palette_lab = cuGlobalConsts.palette_lab;
    int *palette_size = cuGlobalConsts.palette_size;

    PalettePair *palette_pairs = cuGlobalConsts.palette_pairs;
    float *prob_c = cuGlobalConsts.prob_c;

    bool *palette_complete = cuGlobalConsts.palette_complete;


    if (*palette_complete) {
        memcpy(average_palette, palette_lab, (*palette_size) *sizeof(LabColor));
        return;
    }
    for (int i = 0; i < (*palette_size) >> 1; i++) {
        PalettePair pair = palette_pairs[i];
        float weight_a = prob_c[pair.a];
        float weight_b = prob_c[pair.b];
        float total_weight = weight_a + weight_b;
        weight_a /= total_weight;
        weight_b /= total_weight;

        LabColor ca = palette_lab[pair.a];
        LabColor cb = palette_lab[pair.b];

        LabColor avg = {ca.L*weight_a + cb.L*weight_b,
                        ca.a*weight_a + cb.a*weight_b,
                        ca.b*weight_a + cb.b*weight_b};
        
        average_palette[pair.a] = avg;
        average_palette[pair.b] = avg;
    }
    
    }
}

/**
 * @brief Associate Pixels to specific super pixels
 */
__global__ void kernelAssociatetoSuperPixels() {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //TODO: RUN ON ONE KERNAL FOR NOW
    if (index == 0){
    

    // *** TODO TRANSFER OVER CONSTANTS ***//
    int M_pix = cuGlobalConsts.M_pix;
    int S = cuGlobalConsts.S;
    int out_height = cuGlobalConsts.out_height;
    int out_width = cuGlobalConsts.out_width;
    int in_height = cuGlobalConsts.in_height;
    int in_width = cuGlobalConsts.in_width;

    int *region_map = cuGlobalConsts.region_map;
    FloatVec *superPixel_pos = cuGlobalConsts.superPixel_pos;
    int *palette_assign = cuGlobalConsts.palette_assign;
    LabColor *input_img_lab = cuGlobalConsts.input_img_lab;
    LabColor *average_palette = cuGlobalConsts.average_palette;

    //Global bois
    float *distance = new float[M_pix];

    for (int i = 0; i < M_pix; i++) distance[i] = -1.0f;
    
    for (int j = 0; j < out_height; ++j) {
        for (int i = 0; i < out_width; ++i) {
            
            // get local region
            int idx = out_width * j + i;
            FloatVec center = superPixel_pos[idx];
            int min_x = (int) fmaxf(0.0f, center.x - S);
            int min_y = (int) fmaxf(0.0f, center.y - S);
            int max_x = (int) fminf((float) (in_width - 1), center.x + S);
            int max_y = (int) fminf((float) (in_height - 1), center.y + S);            
            //printf("iter %d superpixel %d: (%d, %d) -> (%d, %d)\n", iter, out_width * j + i, min_x, min_y, max_x, max_y);
            int x = (int) round(center.x);
            int y = (int) round(center.y);

            LabColor sp_color = average_palette[palette_assign[idx]];

            // within region
            for (int yy = min_y; yy <= max_y; ++yy) {
                for (int xx = min_x; xx <= max_x; ++xx) {
                    int curr_idx = yy * in_width + xx;

                    // check new distance
                    float dist_new = cuDevDist_k(m_gerstner, S, sp_color.L, sp_color.a, sp_color.b, 
                                            x, y, input_img_lab[curr_idx].L, input_img_lab[curr_idx].a, 
                                            input_img_lab[curr_idx].b, xx, yy);

                    // Check if the distance is less in order to minimize
                    if (distance[curr_idx] < 0 || dist_new < distance[curr_idx]) {
                        distance[curr_idx] = dist_new;
                        region_map[curr_idx] = out_width*j + i;
                    }
                }
            }
        }
    }

    delete[] distance;
    }
}
/**
 * @brief Associate Pixels to specific super pixels
 */
__global__ void kernelSmoothPositions() {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //TODO: RUN ON ONE KERNAL FOR NOW
    if (index == 0){
    

    // *** TODO TRANSFER OVER CONSTANTS ***//
    int N_pix = cuGlobalConsts.N_pix;
    int out_height = cuGlobalConsts.out_height;
    int out_width = cuGlobalConsts.out_width;

    FloatVec *superPixel_pos = cuGlobalConsts.superPixel_pos;
    LabColor *sp_mean_lab = cuGlobalConsts.sp_mean_lab;
    LabColor *buf_lab = cuGlobalConsts.buf_lab;


    // smooth positions
    for (int j = 0; j < out_height; j++) {
        for (int i = 0; i < out_width; i++) {                
            int spidx = j*out_width + i;
            FloatVec sum = {0, 0};
            float count = 0.0f;
            if(i > 0) {
                sum.x += superPixel_pos[j*out_width + i-1].x;
                sum.y += superPixel_pos[j*out_width + i-1].y;
                count += 1.0f;
            }
            if(i < out_width -1) {
                sum.x += superPixel_pos[j*out_width + i+1].x;
                sum.y += superPixel_pos[j*out_width + i+1].y;
                count += 1.0f;
            }
            if(j > 0) {
                sum.x += superPixel_pos[(j-1)*out_width + i].x;
                sum.y += superPixel_pos[(j-1)*out_width + i].y;
                count += 1.0f;
            }
            if(j < out_height - 1) {
                sum.x += superPixel_pos[(j+1)*out_width + i].x;
                sum.y += superPixel_pos[(j+1)*out_width + i].y;
                count += 1.0f;
            }
            sum.x /= count;
            sum.y /= count;
            FloatVec pos = superPixel_pos[spidx];
            FloatVec newPos = {0, 0};
            if(i == 0 || i == out_width -1) {
                newPos.x = pos.x;
            } else {
                newPos.x = (0.55f)*pos.x + 0.45f*sum.x;
            }
            if(j == 0 || j == out_height - 1) {
                newPos.y = pos.y;
            } else {
                newPos.y = 0.55f*pos.y + 0.45f*sum.y;
            }
            // printf("pos: (%f, %f) -> (%f, %f)\n", pos.x, pos.y, newPos.x, newPos.y);
            superPixel_pos[spidx] = newPos;
        }
    }

    // smooth colors
    for(int j = 0; j < out_height; ++j) {
        for(int i = 0; i < out_width; ++i) {

        //get bounds of 3x3 kernel (make sure we don't go off the image)
        int min_x = cuDevMax(0,i-1);
        int max_x = cuDevMin(out_width-1,i+1);
        int min_y = cuDevMax(0,j-1);
        int max_y = cuDevMin(out_height-1,j+1);

        //Initialize
        LabColor sum = {0.f, 0.f, 0.f};
        float weight = 0.f;

        //get current SP color and (grid) position
        LabColor superpixel_color = sp_mean_lab[j*out_width + i];
        //FloatVec p = {(float) j, (float) i};

        //get bilaterally weighted average color of SP neighborhood
        for(int ii = min_x; ii<= max_x; ++ii) {
            for(int jj = min_y; jj<=max_y; ++jj) {
            
            LabColor c_n = sp_mean_lab[jj*out_width + ii];
            float d_color = (float) sqrt(pow(superpixel_color.L - c_n.L, 2.f) +
                                            pow(superpixel_color.a - c_n.a, 2.f) +
                                            pow(superpixel_color.b - c_n.b, 2.f));
            float w_color = cuDevGaussian(d_color, 2.0f ,0.0f);
            float d_pos = (float) sqrt(pow((float) i-ii, 2.f) + pow((float) j-jj, 2.f));
            float w_pos = cuDevGaussian(d_pos, 0.97f, 0.0f);
            float w_total = w_color*w_pos;

            weight += w_total;
            sum.L += c_n.L*w_total;
            sum.a += c_n.a*w_total;
            sum.b += c_n.b*w_total;
            }
        }
        sum.L *= 1.0f/weight;
        sum.a *= 1.0f/weight;
        sum.b *= 1.0f/weight;
        buf_lab[j*out_width + i] = sum;
        }
    }
    
    //update the SP mean colors with the smoothed values
    memcpy(sp_mean_lab, buf_lab, N_pix * sizeof(LabColor));

    }
}
/**
 * @brief Associate Pixels to specific super pixels
 */
__global__ void kernelAssociateToPalette() {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //TODO: RUN ON ONE KERNAL FOR NOW
    if (index == 0){
    

    // *** TODO TRANSFER OVER CONSTANTS ***//
    int N_pix = cuGlobalConsts.N_pix;
    int K_colors = cuGlobalConsts.K_colors;
    int prob_sp = cuGlobalConsts.prob_sp;
    

    int *palette_size = cuGlobalConsts.palette_size;
    float *prob_c_if_sp = cuGlobalConsts.prob_c_if_sp;
    LabColor *sp_mean_lab = cuGlobalConsts.sp_mean_lab;
    LabColor *palette_lab = cuGlobalConsts.palette_lab;
    int *palette_assign = cuGlobalConsts.palette_assign;
    float *prob_c = cuGlobalConsts.prob_c;
    float *T = cuGlobalConsts.T;


    // Allocate local memory
    float *new_prob_c = new float[*palette_size];

    memset(new_prob_c, 0, (*palette_size) * sizeof(float));
    memset(prob_c_if_sp, 0, K_colors * 2 * N_pix *sizeof(float));
    
    // Update superpixel colors from color palette based on P(c_k|p_s) calculation
    for(int p = 0; p < N_pix; p++) {
        // Get the best color value to update the superpixel color
        int best_c = -1;
        float best_norm_val = 0.0f;
        double sum_prob = 0;
        double *probs = new double[*palette_size];

        for (int c = 0; c < (*palette_size); c++){

            // m_s' - c_k TODO: MIGHT NOT WORK?
            LabColor pixDiff;
            pixDiff.L = sp_mean_lab[p].L - palette_lab[c].L;
            pixDiff.a = sp_mean_lab[p].a - palette_lab[c].a;
            pixDiff.b = sp_mean_lab[p].b - palette_lab[c].b;

            // || m_s' - c_k ||
            float norm_val = sqrt(pow(pixDiff.L, 2.f) + pow(pixDiff.a, 2.f) + pow(pixDiff.b, 2.f));

            //  - (|| m_s' - c_k ||/T)
            float pow_val = -1.0f*(norm_val/(*T));
            float prob = prob_c[c] * exp(pow_val);
            
            probs[c] = prob;
            sum_prob += prob;

            //Update if better value
            if (best_c < 0 || norm_val < best_norm_val){
                best_c = c;
                best_norm_val = norm_val;
            }
        } 

        // update palette assignment
        palette_assign[p] = best_c;

        for(int c = 0; c < (*palette_size); c++) {
            double p_norm = probs[c]/sum_prob;
            prob_c_if_sp[c*(N_pix) + p] = p_norm;
            new_prob_c[c] += prob_sp*p_norm;
        }

        delete[] probs;
    }
    
    // update color probabilities
    memcpy(prob_c, new_prob_c, (*palette_size)*sizeof(float));

    // Delete the stuff
    delete[] new_prob_c;

    }
    

}

/**
 * @brief Refine the palette
 */
__global__ void kernelRefinePalette() {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //TODO: RUN ON ONE KERNAL FOR NOW
    if (index == 0){
    

    // *** TODO TRANSFER OVER CONSTANTS ***//
    int N_pix = cuGlobalConsts.N_pix;
    int K_colors = cuGlobalConsts.K_colors;
    int prob_sp = cuGlobalConsts.prob_sp;
    
    int *palette_size = cuGlobalConsts.palette_size;
    float *prob_c_if_sp = cuGlobalConsts.prob_c_if_sp;
    LabColor *sp_mean_lab = cuGlobalConsts.sp_mean_lab;
    float *prob_c = cuGlobalConsts.prob_c;
    LabColor *palette_lab = cuGlobalConsts.palette_lab;
    PalettePair *palette_pairs = cuGlobalConsts.palette_pairs;
    bool *palette_complete = cuGlobalConsts.palette_complete;

    bool *converged = cuGlobalConsts.converged;
    float *T = cuGlobalConsts.T;


    float palette_error = 0.f;

    //TODO: DIFF FROM THERE IMPLEMENTATION? CHECK?
    for (int c = 0; c < (*palette_size); c++){

        LabColor c_sum = {0.0f,0.0f,0.0f};
        // Observe all superpixels to get sum of equation
        for (int p = 0; p < N_pix; p++){
            c_sum.L += sp_mean_lab[p].L * prob_c_if_sp[c*N_pix + p] * prob_sp;
            c_sum.a += sp_mean_lab[p].a * prob_c_if_sp[c*N_pix + p] * prob_sp;
            c_sum.b += sp_mean_lab[p].b * prob_c_if_sp[c*N_pix + p] * prob_sp;
        }
        
        if (prob_c[c] > 0) {
            LabColor last = palette_lab[c];
            //Update palette color
            palette_lab[c].L = c_sum.L/prob_c[c];
            palette_lab[c].a = c_sum.a/prob_c[c];
            palette_lab[c].b = c_sum.b/prob_c[c];
            LabColor curr = palette_lab[c];

            palette_error += sqrt(pow(last.L-curr.L, 2.0f) + pow(last.a-curr.a, 2.0f) + pow(last.b-curr.b, 2.0f));
        }
    }
   
    #ifdef RUN_DEBUG
    printf("expand... ");
    #endif

    if (palette_error < kPaletteErrorTolerance) {
        // check for convergence, lower temperature
        if ((*T) <= kTF) {
            (*converged) = true;
        } else {
            (*T) = fmaxf((*T)*kDT, kTF);
        }
        
        // if palette is incomplete
        if (!(*palette_complete)) {
            int *splits = new int[K_colors];
            
            int curr = 0;
            for (int i = 0; i < (*palette_size) >> 1; i++) {
                #ifdef DEBUG
                // printf("(%d, %d)\n", palette_pairs[i].a, palette_pairs[i].b);
                #endif

                LabColor color_a = palette_lab[palette_pairs[i].a];
                LabColor color_b = palette_lab[palette_pairs[i].b];

                float error = sqrt(pow(color_a.L-color_b.L, 2.0f) + 
                                pow(color_a.a-color_b.a, 2.0f) + 
                                pow(color_a.b-color_b.b, 2.0f));
                // printf("%f, (%f, %f, %f), (%f, %f, %f)\n", error, color_a.L,color_a.a,color_a.b, color_b.L, color_b.a, color_b.b);
                // determine if split or simply perturb 
                if (error > kSubclusterTolerance) {
                    splits[curr] = i;
                    curr ++;
                } else {
                    float value;
                    LabColor majorAxis;
                    getMajorAxis(palette_pairs[i].a, &value, &majorAxis);
                    color_b.L += majorAxis.L * kSubclusterPertubation;
                    color_b.a += majorAxis.a * kSubclusterPertubation;
                    color_b.b += majorAxis.b * kSubclusterPertubation;
                    // printf("perturbed by: (%f, %f, %f)\n", majorAxis.L, majorAxis.a, majorAxis.b);

                    palette_lab[palette_pairs[i].b] = color_b;
                }
            }

            // should sort splits by distance here.
            if (curr > 0) {
                #ifdef RUN_DEBUG
                printf("expanding... %d, %d", (*palette_size), curr);
                #endif
            }

            for (int i = 0; i < curr; i++) {
                splitColor(splits[i]);

                // if full, seal palette
                if ((*palette_size) >= 2 * K_colors) {            
                    #ifdef RUNF_DEBUG
                    printf("COMPLETE\n");                        
                    #endif
                    
                    #ifdef RUN_DEBUG
                    printf("averaging... %d ", *palette_size);
                    #endif

                    // Need to change this at some point
                    inlineGetAveragedPalette();

                    #ifdef RUN_DEBUG
                    printf("averaged... ");
                    #endif

                    inlineCondensePalette();
                    break;
                }
            }       
            delete[] splits;
        }
    }
    }
}

/**
 * @brief Process the output image
 */
__global__ void kernelProcessOutputImage() {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //TODO: RUN ON ONE KERNAL FOR NOW
    if (index == 0){
    

    // *** TODO TRANSFER OVER CONSTANTS ***//
    int out_width = cuGlobalConsts.out_width;
    int out_height = cuGlobalConsts.out_height;
    int in_width = cuGlobalConsts.in_width;
    int in_height = cuGlobalConsts.in_height;

    
    LabColor *average_palette = cuGlobalConsts.average_palette;
    int *palette_assign = cuGlobalConsts.palette_assign;
    unsigned char  *output_img = cuGlobalConsts.output_img;
    unsigned char  *spoutput_img = cuGlobalConsts.spoutput_img;
    int *region_map = cuGlobalConsts.region_map;

    // Create the output image
    for (int j = 0; j < out_height; j++) {
        for (int i = 0; i < out_width; i++) {
            int idx = j*out_width + i;
            LabColor color = average_palette[palette_assign[idx]];

            cuDevlab2rgb(color.L, color.a, color.b, 
                    &(output_img[3*idx]), &(output_img[3*idx + 1]), &(output_img[3*idx + 2]));

        }
    }

    // Create superpixel output image
    for (int j = 0; j < in_height; j++) {
        for (int i = 0; i < in_width; i++) {
            int idx = j*in_width + i;
            if ((region_map[idx]/out_width % 2 == 0 && region_map[idx] % 2 == 0) ||
                (region_map[idx]/out_width % 2 == 1 && region_map[idx] % 2 == 1)) {
                spoutput_img[idx*3] = 0;
                spoutput_img[idx*3 + 1] = 0;
                spoutput_img[idx*3 + 2] = 0;
            } else {
                spoutput_img[idx*3] = 255;
                spoutput_img[idx*3 + 1] = 255;
                spoutput_img[idx*3 + 2] = 255;
            }
        }
    }



    }


}

//********************************************************//
//*******************  PIXEL FUNCTIONS  ******************//
//********************************************************//


PixImage :: PixImage(unsigned char* input_image, int in_w, int in_h, int out_w, int out_h, int K){
    // Store our new variables
    input_img = input_image;
    in_width = in_w;
    in_height = in_h;
    out_width = out_w;
    out_height = out_h;
    K_colors = K;

    // Get value for number of pixels
    M_pix = in_width * in_height;
    N_pix = out_width * out_height;

    // Initialze basic values
    palette_complete = false;
    palette_size = 0; 
    
    #ifdef TIMING
        //Timing variables
        startAllTime=0.f;
        endAllTime=0.f; 
    #endif

    // Init Them Arrays
    input_img = NULL; 
    input_img_lab = NULL;

    output_img = NULL; 
    spoutput_img = NULL; 
    buf_lab = NULL;  

    superPixel_pos = NULL; 
    sp_mean_lab = NULL; 
    region_map = NULL;  

    palette_pairs = NULL;
    palette_assign = NULL; 
    palette_lab = NULL;  
    average_palette = NULL; 

    prob_c = NULL;      
    prob_sp = 0.0f;      
    prob_c_if_sp = NULL;   

    T = 0.0f;  
    S = sqrt(((float) (M_pix))/((float) (N_pix)));  

    // Cuda Device versions of values               
    cuDev_input_img = NULL; 
    cuDev_input_img_lab = NULL;  

    cuDev_output_img = NULL; 
    cuDev_spoutput_img = NULL; 
    cuDev_buf_lab = NULL;        

    cuDev_superPixel_pos = NULL; 
    cuDev_sp_mean_lab = NULL;  
    cuDev_region_map = NULL;     
    
    cuDev_palette_size = NULL;  //single pointer value
    cuDev_palette_pairs = NULL;
    cuDev_palette_assign = NULL;
    cuDev_palette_lab = NULL;
    cuDev_average_palette = NULL; 
    cuDev_palette_complete = NULL;  //single pointer value

    cuDev_prob_c = NULL;         
    cuDev_prob_c_if_sp = NULL; 

    cuDev_T = NULL;  
    cuDev_converged = NULL; 
}


/**
 * @brief Initializes the superPixel_pos array and the region_map array
 */
void PixImage :: initSuperPixels(){
    
    // Intialize size of kernal
    dim3 blockDim(in_width, in_height, 1);
    dim3 gridDim(1,1);


    kernelInitSuperPixels<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();

}

void PixImage :: updateSuperPixelMeans(){
    
    // Intialize size of kernal
    dim3 blockDim(in_height,in_width, 1);
    dim3 gridDim(1,1);


    kernelUpdateSuperPixelMeans<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
}

void PixImage :: getAveragedPalette() {

    // Intialize size of kernal
    dim3 blockDim(1, 1, 1);
    dim3 gridDim(1,1);


    kernelGetAveragedPalette<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();

}

void PixImage :: initVariables(){
    ///*** Allocate Array Space ***///
    input_img_lab = (LabColor *) wrp_calloc(M_pix, sizeof(LabColor));

    superPixel_pos = (FloatVec *) wrp_calloc(N_pix, sizeof(FloatVec)); 
    region_map = (int *) wrp_calloc(M_pix, sizeof(int));

    palette_lab = (LabColor *) wrp_calloc(K_colors * 2 , sizeof(LabColor));
    average_palette = (LabColor *) wrp_calloc(K_colors * 2 , sizeof(LabColor)); 
    palette_size = 0;
    palette_pairs = (PalettePair *) wrp_calloc(K_colors, sizeof(PalettePair));
    palette_assign = (int *) wrp_calloc(N_pix, sizeof(int));
    prob_c = (float *) wrp_calloc(K_colors * 2 , sizeof(float)); 
    prob_c_if_sp = (float *) wrp_calloc(K_colors * 2 * N_pix, sizeof(float));
    prob_sp = 1.0f/(out_width*out_height); 

    buf_lab = (LabColor *) wrp_calloc(N_pix, sizeof(LabColor));
    output_img = (unsigned char *) wrp_malloc(N_pix * 3); 
    spoutput_img = (unsigned char *) wrp_calloc(M_pix*3, sizeof(unsigned char));
    sp_mean_lab = (LabColor *) wrp_calloc(N_pix, sizeof(LabColor)); 

    // Allocate space for Device, place in global for easy access

    hipMalloc(&cuDev_input_img, 3*M_pix*sizeof(unsigned char));
    hipMalloc(&cuDev_input_img_lab, M_pix* sizeof(LabColor));
    hipMalloc(&cuDev_output_img, N_pix * 3);
    hipMalloc(&cuDev_spoutput_img, M_pix*3*sizeof(unsigned char));
    hipMalloc(&cuDev_buf_lab, N_pix*sizeof(LabColor));
    hipMalloc(&cuDev_superPixel_pos, N_pix * sizeof(FloatVec));
    hipMalloc(&cuDev_sp_mean_lab, N_pix * sizeof(LabColor));
    hipMalloc(&cuDev_region_map, M_pix * sizeof(int));
    hipMalloc(&cuDev_palette_pairs, K_colors*sizeof(PalettePair));
    hipMalloc(&cuDev_palette_assign, N_pix *sizeof(int));
    hipMalloc(&cuDev_palette_lab, K_colors * 2 *sizeof(LabColor));
    hipMalloc(&cuDev_average_palette, K_colors * 2 *sizeof(LabColor));
    hipMalloc(&cuDev_prob_c_if_sp, K_colors * 2 * N_pix * sizeof(float));
        
    hipMalloc(&cuDev_palette_size , sizeof(int));
    hipMalloc(&cuDev_palette_complete, sizeof(bool));
    hipMalloc(&cuDev_T , sizeof(float));
    hipMalloc(&cuDev_converged , sizeof(bool));
    
    // Set values to cuda variables (if calloc simply memset)

    hipMemcpy(cuDev_input_img, input_img, 3*M_pix*sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemset(cuDev_input_img_lab, 0, M_pix* sizeof(LabColor));
    hipMemset(cuDev_output_img, 0, N_pix * 3);
    hipMemset(cuDev_spoutput_img, 0, M_pix*3*sizeof(unsigned char));
    hipMemset(cuDev_buf_lab, 0, N_pix*sizeof(LabColor));
    hipMemset(cuDev_superPixel_pos, 0, N_pix * sizeof(FloatVec));
    hipMemset(cuDev_sp_mean_lab, 0, N_pix * sizeof(LabColor));
    hipMemset(cuDev_region_map, 0, M_pix * sizeof(int));
    hipMemset(cuDev_palette_pairs, 0, K_colors*sizeof(PalettePair));
    hipMemset(cuDev_palette_assign,0, N_pix *sizeof(int));
    hipMemset(cuDev_palette_lab, 0, K_colors * 2 *sizeof(LabColor));
    hipMemset(cuDev_average_palette, 0, K_colors * 2 *sizeof(LabColor));
    hipMemset(cuDev_prob_c_if_sp,0, K_colors * 2 * N_pix * sizeof(float));
        
    hipMemset(cuDev_palette_size , 0, sizeof(int));
    hipMemset(cuDev_palette_complete,false, sizeof(bool));
    hipMemset(cuDev_T , 0.0f, sizeof(float));
    hipMemset(cuDev_converged, false, sizeof(bool));
    
    // Initialize parameters in constant memory in order to take 
    // advantge of cuda optimizations. 

    GlobalConstants params;

    // NON pointer values
    params.in_width = in_width;
    params.in_height = in_height;
    params.M_pix = M_pix; 

    params.out_width = out_width;
    params.out_height = out_height;
    params.N_pix = N_pix;

    params.K_colors = K_colors;
    params.palette_size = &palette_size;

    params.prob_sp = prob_sp;

    // Pointer Values
    params.input_img = cuDev_input_img;
    params.input_img_lab = cuDev_input_img_lab;

    params.output_img = cuDev_output_img;
    params.spoutput_img = cuDev_spoutput_img;
    params.buf_lab = cuDev_buf_lab;

    params.superPixel_pos = cuDev_superPixel_pos;
    params.sp_mean_lab = cuDev_sp_mean_lab;
    params.region_map = cuDev_region_map;

    params.palette_pairs = cuDev_palette_pairs;
    params.palette_assign = cuDev_palette_assign;
    params.palette_lab = cuDev_palette_lab;
    params.average_palette = cuDev_average_palette;
    params.palette_complete = cuDev_palette_complete;
    
    params.prob_c = cuDev_prob_c;
    params.prob_c_if_sp = cuDev_prob_c_if_sp;

    params.T = cuDev_T; 
    params.converged = cuDev_converged;
    params.S = S;     
    
    hipMemcpyToSymbol(HIP_SYMBOL(cuGlobalConsts), &params, sizeof(GlobalConstants)); 

}
void PixImage :: initialize(){
    initVariables();

    ///*** Create input_img_lab version ***///
    dim3 blockDim0(1, 1, 1);
    dim3 gridDim0(1,1);

    kernelCreateInputLAB<<<gridDim0, blockDim0>>>();
    hipDeviceSynchronize();

    ///*** Initialize Superpixel Values ***///
    initSuperPixels();
    updateSuperPixelMeans();


    ///*** Initialize Palette Values ***///
    dim3 blockDim1(in_width, in_height, 1);
    dim3 gridDim1(1,1);

    kernelInitPaletteValues<<<gridDim1, blockDim1>>>();
    hipDeviceSynchronize();

}

void PixImage :: runPixelate(){
    
    #ifdef TIMING
    startAllTime = CycleTimer::currentSeconds();
    #endif

    //*** ******************** ***//
    //*** (4.1) INITIALIZATION ***//
    //*** ******************** ***//
    #ifdef TIMING
    startInitializeTime = CycleTimer::currentSeconds();
    #endif
    
    initialize();
    
    #ifdef TIMING
    endInitializeTime = CycleTimer::currentSeconds();
    #endif
    //*** ******************* ***//
    //*** CORE ALGORITHM LOOP ***//
    //*** ******************* ***//

    bool *converged =   (bool *)wrp_malloc(sizeof(bool));
    *converged = false;
    
    int iter = 0;

    // update superpixel segments
    while (!(*converged) && iter < maxIter) {
        
        #ifdef RUN_DEBUG
        printf("iter %d, %f\n", iter, T);
        #endif

        //*** ************************ ***//
        //*** (4.2) REFINE SUPERPIXELS ***//
        //*** ************************ ***//

        ///*** Update boundaries of pixels Associated with super pixels ***///
        
        #ifdef RUN_DEBUG
        printf("average...");
        #endif

        ///*** Get average colors for palette ***///
        getAveragedPalette();
        
        #ifdef RUN_DEBUG
        printf("DONE\n");
        #endif

        #ifdef RUN_DEBUG
        printf("associate...\n");
        #endif

        ///*** Associate to superpixels ***///

        dim3 blockDim0(1, 1, 1);
        dim3 gridDim0(1,1);

        kernelAssociatetoSuperPixels<<<gridDim0, blockDim0>>>();
        hipDeviceSynchronize();

        
        #ifdef RUN_DEBUG 
        printf("udpate means...");
        #endif
        
        updateSuperPixelMeans();

        #ifdef RUN_DEBUG
        printf("DONE\n");
        #endif

        #ifdef RUN_DEBUG
        printf("smooth...");
        #endif
        
        ///*** Smooth positions of Superpixel and pixel ***///

        dim3 blockDim1(1, 1, 1);
        dim3 gridDim1(1,1);

        kernelSmoothPositions<<<gridDim1, blockDim1>>>();
        hipDeviceSynchronize();

        #ifdef RUN_DEBUG
        printf("DONE\n");
        #endif


        //*** ************************************** ***//
        //*** (4.3) ASSOCIATE SUPERPIXELS TO PALETTE ***//
        //*** ************************************** ***//
        #ifdef RUN_DEBUG
        printf("associate...");
        #endif

        dim3 blockDim2(1, 1, 1);
        dim3 gridDim2(1,1);

        kernelAssociateToPalette<<<gridDim2, blockDim2>>>();
        hipDeviceSynchronize();


        #ifdef RUN_DEBUG
        printf("DONE\n");
        #endif

        // //*** ***************************** ***//
        // //*** (4.3) REFINE + EXPAND PALETTE ***//
        // //*** ***************************** ***//

        
        #ifdef RUN_DEBUG
        printf("refine...");
        #endif
        
        dim3 blockDim3(1, 1, 1);
        dim3 gridDim3(1,1);

        kernelRefinePalette<<<gridDim3, blockDim3>>>();
        hipDeviceSynchronize();

        #ifdef RUN_DEBUG
        printf("DONE\n");
        #endif

        //Transfer converged from device (TODO: maybe bug)
        hipMemcpy(converged, cuDev_converged, sizeof(bool), hipMemcpyDeviceToHost);
    
  
        iter ++;
    }

    free(converged);

    //*** ******************** ***//
    //*** PROCESS OUTPUT IMAGE ***//
    //*** ******************** ***//
    // palette_complete = false;
    // Create output image in rgb color values
    getAveragedPalette();


    dim3 blockDim4(1, 1, 1);
    dim3 gridDim4(1,1);

    kernelProcessOutputImage<<<gridDim4, blockDim4>>>();
    hipDeviceSynchronize();

    //Transfer new image stuff from device (TODO: maybe bug)
    hipMemcpy(output_img, cuDev_output_img, N_pix * 3, hipMemcpyDeviceToHost);
    hipMemcpy(spoutput_img, cuDev_spoutput_img, M_pix*3, hipMemcpyDeviceToHost);
    

    #ifdef TIMING
    endAllTime = CycleTimer::currentSeconds();
    #endif


    #ifdef TIMING
    //*** ******************** ***//
    //*** PRINT TIMING RESULTS ***//
    //*** ******************** ***//
    // Print Total Time to run algorithm (not include get image file and create image file)
    printf("Overall: %.3f s\n", 1000.f * (endAllTime - startAllTime));
    
    printf("\t- Initialize: %.3f s\n", 1000.f * (endInitializeTime - startInitializeTime));
    #endif

}

void PixImage :: freeAll(){

    free(input_img_lab);

    free(output_img); 
    free(sp_mean_lab); 

    free(superPixel_pos);
    free(region_map);

    free(palette_lab); 
    free(prob_c); 

    free(buf_lab);
    
}
